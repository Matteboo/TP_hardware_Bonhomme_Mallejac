#include "Header.cuh"


int main(){
    int n = 3;
    int p = n;
    float* M1=(float*)malloc(n*p * sizeof(double));
    float* M2 = (float*)malloc(n * p * sizeof(double));
    float* Mout_cpu = (float*)malloc(n * p * sizeof(double));
    float* Mout_gpu = (float*)malloc(n * p * sizeof(double));
    MatrixInit(M1, n, p);
    MatrixInit(M2, n, p);

    float* d_M1, * d_M2, * d_Mout;
    MatrixMult( M1, M2, Mout_cpu,n);
    hipMalloc((void**)&d_M1, sizeof(float) * n * p);
    hipMalloc((void**)&d_M2, sizeof(float) * n * p);
    hipMalloc((void**)&d_Mout, sizeof(float) * n * p);
    hipMemcpy(d_M1, M1, sizeof(float) * n * p, hipMemcpyHostToDevice);
    hipMemcpy(d_M2, M2, sizeof(float) * n * p, hipMemcpyHostToDevice);
    cudaMatrixMult<<<5,5>>>(d_M1, d_M2, d_Mout, n);
    hipMemcpy(Mout_gpu, d_Mout, sizeof(float) * n * p, hipMemcpyDeviceToHost);
    //MatrixPrint(M1, n, p);
    //MatrixPrint(M2, n, p);
    MatrixPrint(Mout_cpu,  n,  p);
    MatrixPrint(Mout_gpu, n, p);
    return 0;
}

