#include "Header.cuh"


int main(){
    int n = 4;
    float* raw_data=(float*)malloc(n*n * sizeof(double));
    MatrixInit(raw_data, n, n);
    int p = 2;
    int depth = 2;
    float* C1_data = (float*)malloc(p* p * depth * sizeof(double));
    MatrixInit3D_value(C1_data, p, p, depth, 0.5);
    int m = 1;
    float* S1_data = (float*)malloc(m * m * depth* sizeof(double));
    MatrixInit3D_value(S1_data, m, m, depth, 0);
    int kernel_size = 3;
    float* C1_kernel = (float*)malloc(kernel_size * kernel_size * depth * sizeof(double));
    MatrixInit3D(C1_kernel, kernel_size, kernel_size, depth);
    // creation sur le GPU
    float *d_raw_data, *d_C1_data,*d_S1_data, *d_C1_kernel;
    hipMalloc((void**)&d_raw_data,n *n * sizeof(double));
    hipMalloc((void**)&d_C1_data, p * p * depth * sizeof(double));
    hipMalloc((void**)&d_S1_data, m * m * depth * sizeof(double));
    hipMalloc((void**)&d_C1_kernel, kernel_size * kernel_size * depth * sizeof(double));

    //envoie sur le GPU
    hipMemcpy(d_raw_data, raw_data, n * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C1_kernel, C1_kernel, kernel_size * kernel_size * depth * sizeof(double), hipMemcpyHostToDevice);
    //hipMemcpy(d_C1_data, C1_data, p * p * depth * sizeof(double), hipMemcpyHostToDevice);

    //op�rations
    Conv2D<<<5,5>>>(d_raw_data, d_C1_data, d_C1_kernel, p, kernel_size,depth);
    activation_tanh << <5, 5 >> > (d_C1_data, p * p * depth * sizeof(double));
    subsampling2D <<<5, 5 >>> (d_C1_data, d_S1_data, m, 2, depth);


    //renvoie sur le CPU
    hipMemcpy(C1_data, d_C1_data, p * p * depth * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(S1_data, d_S1_data, m * m * depth * sizeof(double), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    MatrixPrint3D(raw_data, n, n, 1);
    MatrixPrint3D(C1_kernel, kernel_size, kernel_size, depth);
    MatrixPrint3D(C1_data, p, p, depth);
    //MatrixPrint3D(S1_data, m, m, depth);
    return 0;
}

