#include "hip/hip_runtime.h"
#include "Header.cuh"


int main(){
    int Nx = 3;
    int Ny = 3;
    int thread = 32;
    int block_x = ceil(Nx + thread - 1) / thread;
    int block_y = ceil(Ny + thread - 1) / thread;
    dim3 THREADS(thread, thread);
    dim3 BLOCKS(block_y, block_x);
    double* M;
    //int N_thread = 1024;
    //int N_block = (N + N_thread) / N_thread;
    M = (double*)malloc(Nx*Ny);
    hiprandState* dev_random;
    hipMalloc((void**)&dev_random, Nx * Ny* thread* thread * sizeof(hiprandState));
    MatrixInit<<<BLOCKS, THREADS >>>(M, Nx, Ny, dev_random);

    MatrixPrint( M,  Nx, Ny);
    return 0;
}

