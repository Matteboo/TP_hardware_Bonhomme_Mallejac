#include "hip/hip_runtime.h"
#include "Header.cuh"


int main(){
    int n = 3;
    int p = n;
    float* M1=(float*)malloc(n*p * sizeof(double));
    float* M2 = (float*)malloc(n * p * sizeof(double));
    float* Mout_cpu = (float*)malloc(n * p * sizeof(double));
    float* Mout_gpu = (float*)malloc(n * p * sizeof(double));
    MatrixInit(M1, n, p);
    MatrixInit(M2, n, p);

    float* d_M1, * d_M2, * d_Mout;
    //clock_t begin_cpu = clock();
    MatrixMult( M1, M2, Mout_cpu,n);
    //clock_t end_cpu = clock();
    
    hipMalloc((void**)&d_M1, sizeof(float) * n * p);
    hipMalloc((void**)&d_M2, sizeof(float) * n * p);
    hipMalloc((void**)&d_Mout, sizeof(float) * n * p);
    hipMemcpy(d_M1, M1, sizeof(float) * n * p, hipMemcpyHostToDevice);
    hipMemcpy(d_M2, M2, sizeof(float) * n * p, hipMemcpyHostToDevice);
    //clock_t begin_gpu = clock();
    cudaMatrixMult<<<5,5>>>(d_M1, d_M2, d_Mout, n);
    //clock_t end_gpu = clock();
    hipMemcpy(Mout_gpu, d_Mout, sizeof(float) * n * p, hipMemcpyDeviceToHost);
    //double time_spent_cpu = (double) (end_cpu-begin_cpu) / CLOCKS_PER_SEC;
    //double time_spent_gpu = (double) (end_gpu-begin_gpu) / CLOCKS_PER_SEC;
    //MatrixPrint(M1, n, p);
    //MatrixPrint(M2, n, p);
    MatrixPrint(Mout_cpu,  n,  p);
    MatrixPrint(Mout_gpu, n, p);
    //printf("Execution time cpu = %f\n", time_spent_cpu);
    //printf("Execution time gpu = %f\n", time_spent_gpu);
    return 0;
}

