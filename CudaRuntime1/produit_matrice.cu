#include "hip/hip_runtime.h"
#include "Header.cuh"

void MatrixInit(float* M, int n, int p) {
	
	for (int i =0; i < n; i += 1) {
		for (int j = 0; j < p; j += 1) {
			int seed = n * j + i;

			M[i + n * j] = (rand() / (double)(RAND_MAX)) * 2 - 1;
		
		}
	}
}


void MatrixInit3D(float* M, int n, int p, int d) {

	for (int i = 0; i < n; i += 1) {
		for (int j = 0; j < p; j += 1) {
			for (int l = 0; l < d; l += 1) {
				M[i + n * j + l * n * p] = (rand() / (double)(RAND_MAX)) * 2 - 1;
			}
		}
	}
}

void MatrixInit3D_value(float* M, int n, int p, int d , float v) {

	for (int i = 0; i < n; i += 1) {
		for (int j = 0; j < p; j += 1) {
			for (int l = 0; l < d; l += 1) {
				M[i + n * j + l * n * p] = v;
			}
		}
	}
}


void MatrixInit_Value(float* M, int n, int p, float v) {

	for (int i = 0; i < n; i += 1) {
		for (int j = 0; j < p; j += 1) {
			M[i + n * j] = v;
		}
	}
}

void MatrixPrint(float* M, int n, int p) {
	int i = 0;
	int j = 0;
	for (i; i < n; i++) {
		for (j = 0; j < p; j++) {
			printf("  %f  ", M[i + n * j]);
		}
		printf("\n");
	}
}

void MatrixAdd(float* M1, float* M2, float* Mout, int n, int p) {
	int i = 0;
	int j = 0;
	for (i; i < n; i++) {
		for (j = 0; j < p; j++) {
			Mout[i + n * j] = M1[i + n * j] + M2[i + n * j];
		}
	}

}



__global__ void cudaMatrixAdd(float* M1, float* M2, float* Mout, int n, int p) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n * p) {
		Mout[i] = M1[i] + M2[i];

	}
}


void MatrixMult(float* M1, float* M2, float* Mout, int n) {
	for (int i = 0; i < n; ++i){
		for (int j = 0; j < n; ++j){
			float tmp = 0.0;
			for (int h = 0; h < n; ++h){
				tmp += M1[i * n + h] * M2[h * n + j];
			}
			Mout[i * n + j] = tmp;
		}
	}
}


__global__ void cudaMatrixMult(float* M1, float* M2, float* Mout, int n) {
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int i = k / n;		//ligne
	int j = k - n * i;	//colonne
	if (k< n * n) {
		float tmp = 0.0;
		for (int h = 0; h < n; ++h) {
			tmp += M1[i * n + h] * M2[h * n + j];
		}
		Mout[i * n + j] = tmp;
	}
}

__global__ void Conv2D(float* M_in, float* M_out, float* kernel,int size_M_out, int size_kernel, int depth) {
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int f = k / depth;										//face de la sortie
	int i = (k-f* size_M_out* size_M_out) / size_M_out;		//ligne de la sortie
	int j = k - size_M_out * i;								//colonne de la sortie
	if (k < size_M_out * size_M_out*depth) {
		int bound = (size_kernel - 1) / 2;
		float tmp = 0.0;
		for (int h = -bound; h < bound +1; ++h) {
			for (int l = -bound; l < bound + 1; l++) {
				tmp += kernel[f* size_kernel * size_kernel +(h+bound) * size_kernel + (l+bound)] * M_in[(i+h) * (size_M_out+ bound) + j+l];
			}
		}
		M_out[f* size_M_out* size_M_out+i * size_M_out + j] = tmp;

	}
}

__global__ void subsampling2D(float* M_in, float* M_out, int size_M_in, int size_window, int depth) {
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int f = k / depth;										//face de la sortie
	int i = (k - f * size_M_out * size_M_out) / size_M_out;		//ligne de la sortie
	int j = k - size_M_out * i;								//colonne de la sortie
	if (k < size_M_out * size_M_out * depth) {
		float tmp = 0.0;
		for (int h = 0; h < size_window; ++h) {
			for (int l = 0; l < size_window; l++) {
				tmp += 1/(size_window* size_window) * M_in[f * size_M_out * size_M_out*size_window * size_window + (i+h)* size_M_out* size_window+j+l];
			}
		}
		M_out[f * size_M_out * size_M_out + i * size_M_out + j] = tmp;

	}
}

void MatrixPrint3D(float* M, int n, int p, int q) {
	int i = 0;
	int j = 0;
	int k = 0;
	for (i; i < n; i++) {
		for (j = 0; j < p; j++) {
			for (k = 0; k < q; k++) {
				printf("  %f  ", M[i + n * j+k*n*p]);
			}
			printf("\n");
		}
		printf("\n");
		printf("\n");
	}
}
