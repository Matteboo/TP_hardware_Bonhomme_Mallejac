#include "hip/hip_runtime.h"
#include "Header.cuh"

void MatrixInit(float* M, int n, int p) {
	
	for (int i =0; i < n; i += 1) {
		for (int j = 0; j < n; j += 1) {
			int seed = n * j + i;

			M[i + n * j] = (rand() / (double)(RAND_MAX)) * 2 - 1;
		
		}
	}
}

void MatrixPrint(float* M, int n, int p) {
	int i = 0;
	int j = 0;
	for (i; i < n; i++) {
		for (j = 0; j < p; j++) {
			printf("  %f  ", M[i + n * j]);
		}
		printf("\n");
	}
}

void MatrixAdd(float* M1, float* M2, float* Mout, int n, int p) {
	int i = 0;
	int j = 0;
	for (i; i < n; i++) {
		for (j = 0; j < p; j++) {
			Mout[i + n * j] = M1[i + n * j] + M2[i + n * j];
		}
	}

}



__global__ void cudaMatrixAdd(float* M1, float* M2, float* Mout, int n, int p) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n * p) {
		Mout[i] = M1[i] + M2[i];

	}
}


void MatrixMult(float* M1, float* M2, float* Mout, int n) {
	for (int i = 0; i < n; ++i){
		for (int j = 0; j < n; ++j){
			float tmp = 0.0;
			for (int h = 0; h < n; ++h){
				tmp += M1[i * n + h] * M2[h * n + j];
			}
			Mout[i * n + j] = tmp;
		}
	}
}


__global__ void cudaMatrixMult(float* M1, float* M2, float* Mout, int n) {
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int i = k / n;		//ligne
	int j = k - n * i;	//colonne
	if (i < n * n) {
		float tmp = 0.0;
		for (int h = 0; h < n; ++h) {
			tmp += M1[i * n + h] * M2[h * n + j];
		}
		Mout[i * n + j] = tmp;
	}
}


