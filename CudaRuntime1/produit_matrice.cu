#include "hip/hip_runtime.h"
#include "Header.cuh"

void MatrixInit(float* M, int n, int p) {
	
	for (int i =0; i < n; i += 1) {
		for (int j = 0; j < p; j += 1) {
			

			M[i + n * j] = (rand() / (double)(RAND_MAX)) * 2 - 1;
		
		}
	}
}


void MatrixInit3D(float* M, int n, int p, int d) {

	for (int i = 0; i < n; i += 1) {
		for (int j = 0; j < p; j += 1) {
			for (int l = 0; l < d; l += 1) {
				M[i + n * j + l * n * p] = (rand() / (double)(RAND_MAX)) * 2 - 1;
			}
		}
	}
}

void MatrixInit3D_value(float* M, int n, int p, int d , float v) {
	// fill the matrix with the value v
	for (int i = 0; i < n; i += 1) {
		for (int j = 0; j < p; j += 1) {
			for (int k = 0; k < d; k += 1) {
				M[i + n * j + k * n * p] = v;
			}
		}
	}
}


void MatrixInit_Value(float* M, int n, int p, float v) {

	for (int i = 0; i < n; i += 1) {
		for (int j = 0; j < p; j += 1) {
			M[i + n * j] = v;
		}
	}
}

void MatrixPrint(float* M, int n, int p) {
	int i = 0;
	int j = 0;
	for (i; i < n; i++) {
		for (j = 0; j < p; j++) {
			printf("  %f  ", M[i + n * j]);
		}
		printf("\n");
	}
}

void MatrixAdd(float* M1, float* M2, float* Mout, int n, int p) {
	int i = 0;
	int j = 0;
	for (i; i < n; i++) {
		for (j = 0; j < p; j++) {
			Mout[i + n * j] = M1[i + n * j] + M2[i + n * j];
		}
	}

}

__global__ void cudaMatrixAdd(float* M1, float* M2, float* Mout, int n, int p) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n * p) {
		Mout[i] = M1[i] + M2[i];

	}
}

void MatrixMult(float* M1, float* M2, float* Mout, int n) {
	for (int i = 0; i < n; ++i){
		for (int j = 0; j < n; ++j){
			float tmp = 0.0;
			for (int h = 0; h < n; ++h){
				tmp += M1[i * n + h] * M2[h * n + j];
			}
			Mout[i * n + j] = tmp;
		}
	}
}

__global__ void cudaMatrixMult(float* M1, float* M2, float* Mout, int n) {
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int i = k / n;		//ligne
	int j = k - n * i;	//colonne
	if (k< n * n) {
		float tmp = 0.0;
		for (int h = 0; h < n; ++h) {
			tmp += M1[i * n + h] * M2[h * n + j];
		}
		Mout[i * n + j] = tmp;
	}
}

__global__ void Conv2D(float* M_in, float* M_out, float* kernel,int size_M_out, int size_kernel, int depth) {
	int k = blockIdx.x * blockDim.x + threadIdx.x;

	int f = k / (size_M_out * size_M_out);						//face de la sortie
	int i = (k - f * size_M_out * size_M_out) / size_M_out;		//ligne de la sortie
	int j = k - f * size_M_out * size_M_out - size_M_out * i;	//colonne de la sortie
	//k=f*siz*size+i*size+j
	if (k < size_M_out * size_M_out*depth) {
		//printf("k=  %u     f=%u       i= %u      j= %u \n", k, f, i, j);
		int bound = (size_kernel - 1) / 2;
		float tmp = 0.0;
		for (int h = 0; h < size_kernel; h++) {//indice pour parcourir les lignes
			for (int l = 0; l < size_kernel; l++) {// indice pour parcourir les colonnes

				tmp += kernel[f* size_kernel * size_kernel +h * size_kernel + l] * M_in[(i+h)*(size_M_out+ 2*bound) + j+l];
			}
		}
		M_out[f* size_M_out* size_M_out+i * size_M_out + j] = tmp;
	}
}

__global__ void subsampling2D(float* M_in, float* M_out, int size_M_out, int size_window, int depth) {
	int k = blockIdx.x * blockDim.x + threadIdx.x;

	int f = k / (size_M_out * size_M_out);						//face de la sortie
	int i = (k - f * size_M_out * size_M_out) / size_M_out;		//ligne de la sortie
	int j = k - f * size_M_out * size_M_out - size_M_out * i;	//colonne de la sortie
	//k=f*siz*size+i*size+j
	//printf("k=  %u     f=%u       i= %u      j= %u \n", k,f,i,j);

	if (k < size_M_out * size_M_out * depth) {
		//printf("k=  %u     f=%u       i= %u     j= %u \n", k, f, i, j);
		double tmp = 0.0; // variable pour stocker la valeur finale
		for (int h = 0; h < size_window; ++h) {
			for (int l = 0; l < size_window; l++) {
				tmp +=M_in[f * size_M_out * size_M_out*size_window * size_window + (i+h)* size_M_out* size_window+j+l]/(size_window* size_window);
				//printf("index= %u  M[]=%f\n ", f * size_M_out * size_M_out * size_window * size_window + (i + h) * size_M_out * size_window + j + l, M_in[f * size_M_out * size_M_out * size_window * size_window + (i + h) * size_M_out * size_window + j + l]);
			}
		}
		M_out[f * size_M_out * size_M_out + i * size_M_out + j] = tmp;
		

	}
	
}

void MatrixPrint3D(float* M, int n, int p, int q) {
	int i = 0;
	int j = 0;
	int k = 0;
	for (k = 0; k < q; k++) {
		for (j = 0; j < p; j++) {
			for (i = 0; i < n; i++)			 {
				printf("  %f  ", M[i + n * j+k*n*p]);
			}
			printf("\n");
		}
		printf("\n");//double saut de ligne quand on change de face
		printf("\n");
	}
}

__global__ void activation_tanh(float* M, int size) {
	//applique tanh � tout les �l�ment de la matrice M
	//Size est le nombre d'ELEMENTS de la matrice M (ligne*colonne*profondeur)
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	if (k < size) {
		M[k] = tanh(M[k]);
	}

}