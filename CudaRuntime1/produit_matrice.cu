#include "hip/hip_runtime.h"
#include "Header.cuh"

__global__ void MatrixInit(double* M, int n, int p, hiprandState* states) {
	
	for (int i = (threadIdx.x + blockIdx.x * blockDim.x); i < n; i += (blockDim.x * gridDim.x)) {
		for (int j = (threadIdx.y + blockIdx.y * blockDim.y); j < p; j += (blockDim.y * gridDim.y)) {
			int seed = n * j + i;

			hiprand_init(seed, i, 0, &states[n * j + i]);
			M[i + n * j] = 1;
				//hiprand_uniform(&states[n * j + i]);
		
		}
	}
}

void MatrixPrint(double* M, int Nx, int Ny) {
	printf("coucou \n");
	int i = 0;
	int j = 0;
	for (i; i < Nx; i++) {
		for (j = 0; j < Ny; j++) {
			printf("  %f  ", M[i + Nx * j]);
		}
		printf("\n");
	}
}