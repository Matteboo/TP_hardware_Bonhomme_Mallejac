#include "hip/hip_runtime.h"
#include "Header.cuh"




__global__ void vector_add(float *out, float *a, float *b, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Handling arbitrary vector size
    if (tid < n){
        out[tid] = a[tid] + b[tid];
    }
}
